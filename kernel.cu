#include "hip/hip_runtime.h"
﻿#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

int main()
{
    bool failed = false;
    for (int i = 1; i <= std::numeric_limits<uint32_t>::max(); ++i)
    {
        thrust::host_vector<uint32_t> temp(i, 1);

        thrust::device_vector<uint32_t> d0 = temp;
        thrust::device_vector<uint32_t> result(d0.size(), 0);

        thrust::inclusive_scan(d0.begin(), d0.end(), result.begin());

        if (result.back() != i)
        {
            if (failed)
                continue;

            std::cout << "at iteration: " << i << "; " << result.back() << " != " << i << std::endl;

            std::cout << "[" << i << " - ";
            failed = true;
        }
        else if (failed)
        {
            std::cout << i << "]\n";
            failed = false;
        }
    }
    return 0;
}